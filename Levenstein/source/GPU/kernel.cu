#include "hip/hip_runtime.h"
#include "kernel.cuh"

#define BLOCK_SIZE 32
#define WARP_SIZE min(BLOCK_SIZE, warpSize)
#define WARPS_PER_BLOCK (BLOCK_SIZE + WARP_SIZE - 1) / WARP_SIZE 

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

bool strings_are_swapped = false;

// device arrays
int* dev_distances_glob = nullptr;
int* dev_accessory_glob = nullptr;

// device strings
char* dev_pattern_glob = nullptr;
char* dev_text_glob = nullptr;

// alphabet variables
char alphabet_first_symbol_glob;
char alphabet_size_glob;

string pattern_glob;
string text_glob;

// error checking
void checkCUDAError(const char* msg, int line = -1) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		if (line >= 0) {
			fprintf(stderr, "Line %d: ", line);
		}
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

// function which was only used to debug the code
// it copies a device 2D array to host and print it in specified mode
// which describes whether the array is stored in the column or the row order
void printPartOfDevArray(void* dev_arr, int byte_size, int type, int mode = ROW_MODE, int row_size = -1)
{
	if (!dev_arr || byte_size < 0 || row_size < -1) return;

	void* host_void_array = nullptr;
	int word_length = 1;
	switch (type)
	{
	case INT_ARR:
		word_length = sizeof(int);
		break;
	case CHAR_ARR:
		word_length = sizeof(char);
		break;
	default:
		return;
	}

	host_void_array = malloc(byte_size);
	if (!host_void_array)
		return;

	hipMemcpy(host_void_array, dev_arr, byte_size, hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("hipMemcpy host_void_array");

	int words_count = byte_size / word_length;
	int rows = row_size == -1 ? words_count : words_count / row_size;
	int columns = row_size == -1 ? words_count : row_size;
	int k = 0;
	for (int i = 0; i < rows; i++)
	{
		if (mode == COL_MODE)
			k = i;
		for (int j = 0; j < columns && k < words_count; j++)
		{
			switch (type)
			{
			case INT_ARR:
				printf("%d ", ((int*)host_void_array)[k]);
				break;
			case CHAR_ARR:
				printf("%c ", ((char*)host_void_array)[k]);
				break;
			default:
				return;
			}
			if (mode == COL_MODE)
			{
				k += rows;
			}
			else
			{
				k++;
			}
		}
		printf("\n");
	}
	fflush(stdout);
	free(host_void_array);
}

// function that initializes all necessary memory on the gpu
bool LevenshteinGPU::init(string& first, string& second, char alphabet_first_symbol, char alphabet_last_symbol)
{
	alphabet_first_symbol_glob = alphabet_first_symbol;
	alphabet_size_glob = alphabet_last_symbol - alphabet_first_symbol + 1;

	// swapping strings to increase number of working threads
	strings_are_swapped = first.size() > second.size();
	if (first.size() <= second.size())
	{
		pattern_glob = string(first);
		text_glob = string(second);
	} 
	else
	{
		pattern_glob = string(second);
		text_glob = string(first);
	}

	int pattern_size = pattern_glob.size();
	int text_size = text_glob.size();
	int array_size = (pattern_size + 1) * (text_size + 1);
	

	// allocating buffers
	hipMalloc((void**)&dev_distances_glob, array_size * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc error");

	hipMalloc((void**)&dev_pattern_glob, pattern_size * sizeof(char));
	checkCUDAErrorWithLine("hipMalloc error");

	hipMalloc((void**)&dev_text_glob, text_size * sizeof(char));
	checkCUDAErrorWithLine("hipMalloc error");

	hipMalloc((void**)&dev_accessory_glob, alphabet_size_glob * (text_size + 1) * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc error");

	// coping strings into GPU memory
	hipMemcpy((void*)dev_pattern_glob, (void*)pattern_glob.c_str(), pattern_size * sizeof(char), hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("hipMemcpy error");

	hipMemcpy((void*)dev_text_glob, (void*)text_glob.c_str(), text_size * sizeof(char), hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("hipMemcpy error");

	return true;
}

// kernel which computes X array
__global__ void kernFillAccessoryArray(int alphabet_size, int text_size, char* text, int* accessory_array, char alphabet_first_symbol)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < alphabet_size)
	{
		int i = index * (text_size + 1);
		int j = 0;
		char symbol = alphabet_first_symbol + index;

		accessory_array[i] = 0;
		i++;

		while (j < text_size)
		{
			accessory_array[i] = text[j] == symbol ? j + 1 : accessory_array[i - 1];
			i++;
			j++;
		}		
	}
}

// part of kernel which implements determining the value of currently computed array element
// based on precalculated Avar, Bvar, etc.
// it is basically extracted code from kernLevenshteinSingleRow
__device__ void computeLevenshteinCell(int text_size, char* pattern, char* text,
	int* distances, int* accessory_array, int iteration, int index, char alphabet_first_symbol,
	int i, int Avar, int Bvar)
{
	// computing values of l and X[l,j]
	int l = pattern[iteration - 1] - alphabet_first_symbol;
	int X = accessory_array[l * (text_size + 1) + index];

	// case of first column 
	if (index == 0)
	{
		distances[i] = iteration;
	}
	// the rest of array
	else
	{
		// case when currently processed symbols of pattern and text are equal
		if (pattern[iteration - 1] == text[index - 1])
		{
			distances[i] = Avar;
		}
		else
		{
			// case when X[l,j] == 0
			if (X == 0)
			{
				distances[i] = min(min(Avar, Bvar), iteration + index - 1) + 1;
			}
			else
			{
				int Cvar = distances[(iteration - 1) * (text_size + 1) + X - 1];
				distances[i] = min(min(Avar, Bvar), Cvar + index - 1 - X) + 1;
			}
		}
	}
}

// function that releases whole allocated memory
void LevenshteinGPU::end()
{
	hipFree(dev_distances_glob);
	hipFree(dev_accessory_glob);
	hipFree(dev_pattern_glob);
	hipFree(dev_text_glob);
}

// kernel that performs computation of single row in D array
__global__ void kernLevenshteinSingleRow(int pattern_size, int text_size,
	char* pattern, char* text,
	int* distances, int* accessory_array,
	char alphabet_first_symbol, int iteration)
{
	// thread index
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index <= text_size && iteration <= pattern_size) // iteration represents i and index represents j in the article
	{
		int i = iteration * (text_size + 1) + index;

		if (iteration == 0)
		{
			distances[i] = index;
			return;
		}

		int Avar = 0;
		if(index > 0)
			Avar = distances[i - (text_size + 1) - 1];
		int Bvar = distances[i - (text_size + 1)];

		computeLevenshteinCell(text_size, pattern, text, distances, accessory_array,
			iteration, index, alphabet_first_symbol, i, Avar, Bvar);
	}
}

// function that computes X array described in the article
void fillAccessoryArray()
{
	int accessory_number_of_blocks = (alphabet_size_glob + BLOCK_SIZE) / BLOCK_SIZE;
	kernFillAccessoryArray << < accessory_number_of_blocks, BLOCK_SIZE >> > (alphabet_size_glob, text_glob.size(), dev_text_glob, dev_accessory_glob, alphabet_first_symbol_glob);
	checkCUDAErrorWithLine("kernComputeLevenshteinArray");

	hipDeviceSynchronize();
	checkCUDAErrorWithLine("synchronization");
}

// function that copies the last element in computed distance array form GPU
// and returns it 
int getResult()
{
	int result = -1;
	hipMemcpy(&result, &(dev_distances_glob[(pattern_glob.size() + 1) * (text_glob.size() + 1) - 1]), sizeof(int), hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("copying result shared and shuffle");
	return result;
}

// function that computes Levenshtein distance with shared memory
int singleRow()
{
	int number_of_blocks = (text_glob.size() + BLOCK_SIZE) / BLOCK_SIZE;
	for (int i = 0; i <= pattern_glob.size(); i++)
	{
		kernLevenshteinSingleRow << <number_of_blocks, BLOCK_SIZE >> > (pattern_glob.size(), text_glob.size(),
			dev_pattern_glob, dev_text_glob, dev_distances_glob, dev_accessory_glob, alphabet_first_symbol_glob, i);
		checkCUDAErrorWithLine("shared memory kernel");

		hipDeviceSynchronize();
		checkCUDAErrorWithLine("synchronization");
	}

	return getResult();
}


// computing Levenshtein distance by the use of sepecified algorithm
int LevenshteinGPU::computeLevenshteinDistance()
{
	fillAccessoryArray();

	return singleRow();
}

// mapping 2D indices to 1D index
int indicesToIndex(int i, int j, int row_size)
{
	return i * row_size + j;
}

// generating code for transformation
string getTransformationCodeGPU(int i, int j, int move)
{
	stringstream stream;
	
	switch (move)
	{
	case GPU_MOVE_CHANGE:
		stream << "C:" << i - 1 << ":" << j - 1;
		break;
	case GPU_MOVE_ADD:
		stream << "I:" << i << ":" << j - 1;
		break;
	case GPU_MOVE_DELETE:
		stream << "R:" << i - 1;
		break;
	default:
		break;
	}
	stream << endl;
	return stream.str();
}

// function that extracts transformations from D array
stringstream LevenshteinGPU::extractTransformations()
{
	int* distances = new int[(pattern_glob.size() + 1) * (text_glob.size() + 1)];
	stringstream sstream = stringstream();
	if (!distances)
		return sstream;
	int i = pattern_glob.size();
	int j = text_glob.size();
	while (i > -1 && j > -1)
	{
		if (i > 0)
		{
			// case when change operation is allowed
			if (j > 0)
			{
				// case when the symbols of both strings are the same
				if (pattern_glob[i - 1] == text_glob[j - 1])
				{
					// nothing to transform
					i--;
					j--;
				}
				else
				{
					// indices are mapped as follows:
					// i-1,j-1 -> ind11
					// i-1,j -> ind10
					// i,j-1 -> ind01
					int ind11 = indicesToIndex(i - 1, j - 1, text_glob.size() + 1);
					int ind10 = indicesToIndex(i - 1, j, text_glob.size() + 1);
					int ind01 = indicesToIndex(i, j - 1, text_glob.size() + 1);
					if (distances[ind11] <= distances[ind10])
					{
						// case when D[i-1,j-1] is the minimal value among the considered ones
						if (distances[ind11] <= distances[ind01])
						{
							// depending on swapping the change transformation is performed from pattern or from text
							if (!strings_are_swapped)
							{

								sstream << getTransformationCodeGPU(i, j, GPU_MOVE_CHANGE);
							}
							else
							{
								sstream << getTransformationCodeGPU(j, i, GPU_MOVE_CHANGE);
							}
							i--;
							j--;
						}
						// case when D[i,j-1] is the minimal value among the considered ones
						else
						{
							// depending on swapping the change transformation is performed from pattern or from text
							if (!strings_are_swapped)
							{
								sstream << getTransformationCodeGPU(i, j, GPU_MOVE_ADD);
							}
							else
							{
								sstream << getTransformationCodeGPU(j, i, GPU_MOVE_DELETE);
							}
							j--;
						}
					}
					else
					{
						// case when D[i-1,j] is the minimal value among the considered ones
						if (distances[ind10] < distances[ind01])
						{
							// depending on swapping the change transformation is performed from pattern or from text
							if (!strings_are_swapped)
							{
								sstream << getTransformationCodeGPU(i, j, GPU_MOVE_DELETE);
							}
							else
							{
								sstream << getTransformationCodeGPU(j, i, GPU_MOVE_ADD);
							}
							i--;
						}
						// case when D[i,j-1] is the minimal value among the considered ones
						else
						{
							// depending on swapping the change transformation is performed from pattern or from text
							if (!strings_are_swapped)
							{
								sstream << getTransformationCodeGPU(i, j, GPU_MOVE_ADD);
							}
							else
							{
								sstream << getTransformationCodeGPU(j, i, GPU_MOVE_DELETE);
							}
							j--;
						}
					}
				}
			}
			// case when change operation is not allowed
			else
			{
				// depending on swapping the change transformation is performed from pattern or from text
				if (!strings_are_swapped)
				{
					sstream << getTransformationCodeGPU(i, j, GPU_MOVE_DELETE);
				}
				else
				{
					sstream << getTransformationCodeGPU(j, i, GPU_MOVE_ADD);
				}
				i--;
			}
		}
		// case when change operation is not allowed
		else
		{
			// depending on swapping the change transformation is performed from pattern or from text
			if (j > 0)
			{
				if (!strings_are_swapped)
				{
					sstream << getTransformationCodeGPU(i, j, GPU_MOVE_ADD);
				}
				else
				{
					sstream << getTransformationCodeGPU(j, i, GPU_MOVE_DELETE);
				}
			}
			j--;
		}
	}
	delete distances;
	return sstream;

}